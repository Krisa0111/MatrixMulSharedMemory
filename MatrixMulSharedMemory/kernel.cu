#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 16
#define TILE_SIZE 4

__global__ void matrixMultiplication(int* a, int* b, int* c, int n) {

    __shared__ int s_a[TILE_SIZE][TILE_SIZE];
    __shared__ int s_b[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    int sum = 0;
    for (int i = 0; i < n / TILE_SIZE; i++) {
        s_a[ty][tx] = a[row * n + i * TILE_SIZE + tx];
        s_b[ty][tx] = b[(i * TILE_SIZE + ty) * n + col];
        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++) {
            sum += s_a[ty][j] * s_b[j][tx];
        }
        __syncthreads();
    }
    c[row * n + col] = sum;
}

void printMatrix(int* m, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", m[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {

    int a[N * N], b[N * N], c[N * N], d[N * N];
    int* dev_a, * dev_b, * dev_c;

    hipMalloc((void**)&dev_a, N * N * sizeof(int));
    hipMalloc((void**)&dev_b, N * N * sizeof(int));
    hipMalloc((void**)&dev_c, N * N * sizeof(int));

    for (int i = 0; i < N * N; i++) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
        d[i] = 0;
    }

    printf("Input matrices:\n");
    printf("Matrix A:\n");
    printMatrix(a, N);
    printf("Matrix B:\n");
    printMatrix(b, N);

    hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(N / TILE_SIZE, N / TILE_SIZE);
    dim3 block(TILE_SIZE, TILE_SIZE);

    matrixMultiplication << <grid, block >> > (dev_a, dev_b, dev_c, N);

    hipMemcpy(c, dev_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Output matrix:\n");
    printMatrix(c, N);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            for (int k = 0; k < N; k++) {
                d[i * N + j] += a[i * N + k] * b[k * N + j];
            }
        }
    }

    printf("Expected output matrix:\n");
    printMatrix(d, N);

    for (int i = 0; i < N * N; i++) {
        if (c[i] != d[i]) {
            printf("Error: matrix multiplication result does not match expected result.\n");
            break;
        }
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

